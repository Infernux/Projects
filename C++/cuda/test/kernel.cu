
#include <hip/hip_runtime.h>
__global__ void kadd(float *a, float *b, float *c, const unsigned int el_per_thread)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int offset = i * el_per_thread;

  for(unsigned int idx = 0; idx < el_per_thread; idx++) {
    c[offset+idx] = a[offset+idx] + b[offset+idx];
  }
}

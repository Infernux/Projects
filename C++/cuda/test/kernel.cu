
#include <hip/hip_runtime.h>
__global__ void kadd(float *a, float *b, float *c)
{
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

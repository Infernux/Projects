#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#include <hip/hip_runtime.h>

#include "kernel.cuh"

#define N 10

void randomizeFloatArray(float *arr, const uint32_t arr_size, const uint32_t seed)
{
  srand(seed);
  for(uint32_t i = 0; i < arr_size; ++i)
  {
    arr[i] = rand();
  }
}

void printFloatArray(float *arr, const uint32_t arr_size)
{
  for(uint32_t i = 0; i < arr_size; ++i)
  {
    printf("%f, ", arr[i]);
  }
  printf("\n");
}

int main()
{
  float a[N], b[N], c[N];
  float *c_in_a, *c_in_b, *c_out_c;

  randomizeFloatArray(a, N, 42);
  randomizeFloatArray(b, N, 43);

  printFloatArray(a, N);
  printFloatArray(b, N);

  hipMalloc((void **) &c_in_a, N * sizeof(float));
  hipMalloc((void **) &c_in_b, N * sizeof(float));
  hipMalloc((void **) &c_out_c, N * sizeof(float));

  hipMemcpy(c_in_a, a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(c_in_b, b, N * sizeof(float), hipMemcpyHostToDevice);

  kadd<<<1, N>>>(c_in_a,c_in_b,c_out_c);

  hipMemcpy(c, c_out_c, N * sizeof(float), hipMemcpyDeviceToHost);

  printFloatArray(c, N);

  hipFree(c_in_a);
  hipFree(c_in_b);
  hipFree(c_out_c);
}

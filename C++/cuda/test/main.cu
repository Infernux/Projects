#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#include <hip/hip_runtime.h>

#include "kernel.cuh"

#define N 100000
#define BLOCK_SIZE 256

void compareBuffers(const float *a, const float *b, const uint32_t arr_size)
{
  uint32_t total_failed = 0;
  for(uint32_t i = 0; i < arr_size; ++i)
  {
    if(a[i] != b[i]) {
      printf("Failed index %d\n", i);
      total_failed++;
    }
  }
  printf("failed (%d/%d)\n", total_failed, arr_size);
}

void referenceAdd(const float *a, const float *b, float *c, const uint32_t arr_size)
{
  for(uint32_t i = 0; i < arr_size; ++i)
  {
    c[i] = a[i] + b[i];
  }
}

void randomizeFloatArray(float *arr, const uint32_t arr_size, const uint32_t seed)
{
  srand(seed);
  for(uint32_t i = 0; i < arr_size; ++i)
  {
    arr[i] = rand();
  }
}

void initSimpleFloatArray(float *arr, const uint32_t arr_size, const uint32_t offset)
{
  for(uint32_t i = 0; i < arr_size; ++i)
  {
    arr[i] = i + offset;
  }
}

void printFloatArray(float *arr, const uint32_t arr_size)
{
  for(uint32_t i = 0; i < arr_size; ++i)
  {
    printf("%f, ", arr[i]);
  }
  printf("\n");
}

int main()
{
  float *a, *b, *c, *ref_c;
  float *c_in_a, *c_in_b, *c_out_c;

  a = (float*)malloc(sizeof(float) * N);
  b = (float*)malloc(sizeof(float) * N);
  c = (float*)malloc(sizeof(float) * N);
  ref_c = (float*)malloc(sizeof(float) * N);

  initSimpleFloatArray(a, N, 0);
  initSimpleFloatArray(b, N, 43);

  //printFloatArray(a, N);
  //printFloatArray(b, N);

  hipMalloc((void **) &c_in_a, N * sizeof(float));
  hipMalloc((void **) &c_in_b, N * sizeof(float));
  hipMalloc((void **) &c_out_c, N * sizeof(float));

  hipMemcpy(c_in_a, a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(c_in_b, b, N * sizeof(float), hipMemcpyHostToDevice);

  //const uint32_t el_per_thread = (uint32_t)ceil((double)N / (double)BLOCK_SIZE);
  const uint32_t el_per_thread = 1;
  const uint32_t block_count = (uint32_t)ceil(((double)N / el_per_thread) / (double)BLOCK_SIZE);

  printf("el per thread : %d, block_count : %d\n", el_per_thread, block_count);

  kadd<<<block_count, BLOCK_SIZE>>>(c_in_a, c_in_b, c_out_c, el_per_thread);

  hipMemcpy(c, c_out_c, N * sizeof(float), hipMemcpyDeviceToHost);

  //printFloatArray(c, N);

  hipFree(c_in_a);
  hipFree(c_in_b);
  hipFree(c_out_c);

  referenceAdd(a, b, ref_c, N);
  compareBuffers(ref_c, c, N);

  free(a);
  free(b);
  free(c);
  free(ref_c);
}
